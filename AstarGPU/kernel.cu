#include "hip/hip_runtime.h"
﻿// Astar.cpp : This file contains the 'main' function. Program execution begins and ends there.
//

#include "hip/hip_runtime.h"
#include ""

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <algorithm>
using namespace std;

struct Edge
{
    double weight;//waga prawdziwa + heurystyczna
    double weightFromStart;//waga od startu
    int idOfNode;//id noda z ktorego ide
    int idOfPreviusNode;//id do ktorego ide

    bool operator < (const Edge& e) const
    {
        return weight < e.weight;
    }

};

struct Coord
{
    int x;
    int y;
};

struct Node
{
    string name;
    int numberOfEdges;
    int* idsOfNeighbor;//tablica z id'kami sasiadow
    double* weightOfEdges;//tablica z wagami krawedzi do sasiadow
};

int getIndexOfNodeByName(vector<string> namesOfNodes, string name);
double calcHeuristicValue(int x1, int y1, int x2, int y2);
double calcHeuristicValue(int x1, int y1);
double getWeightFromStartForNode(vector<Edge> edges, int idOfNode);
bool isEdgeIsOpened(vector<Edge> edges, int idOfNode);
int getIndexOfEdge(vector<Edge> edges, int idOfNode);

__global__ void calcHeuristicValues(int* x, int* y, double* c, int* finishX, int* finishY, int* numberOfPoints);

int main(int argc, char* argv[])
{
    ////DEKLARACJE 

    int numberOfNodes;//licba przypadkow
    vector<string> namesOfNodes;//nazwy - etykiety nodow
    Coord* coords;
    vector<Node> nodes;//wektor z node'ami
    vector<Edge> openEdges;//wekstor z otwartymi akutalnie krawedziami
    vector<Edge> checkedEdges;//sprawdzone krawedzie

    ////CZYTANIE Z PLIKU

    ifstream inputFile(argv[1]);
    string inputString;

    if (inputFile.is_open())
    {
        inputFile >> inputString;
        numberOfNodes = stoi(inputString);

        coords = new Coord[numberOfNodes];

        for (int i = 0; i < numberOfNodes; i++)
        {
            Node tmpNode;
            inputFile >> inputString;
            namesOfNodes.push_back(inputString);
            tmpNode.name = inputString;
            inputFile >> inputString;
            coords[i].x = stoi(inputString);
            inputFile >> inputString;
            coords[i].y = stoi(inputString);
            inputFile >> inputString;
            tmpNode.numberOfEdges = stoi(inputString);
            tmpNode.idsOfNeighbor = new int[tmpNode.numberOfEdges];
            tmpNode.weightOfEdges = new double[tmpNode.numberOfEdges];

            for (int j = 0; j < tmpNode.numberOfEdges; j++)
            {
                inputFile >> inputString;
                tmpNode.idsOfNeighbor[j] = stoi(inputString);
                inputFile >> inputString;
                tmpNode.weightOfEdges[j] = stoi(inputString);
            }

            nodes.push_back(tmpNode);
        }
    }
    else
    {
        cout << "File not found. Run program witch parameters: astarInputPoints.txt S G";
        coords = new Coord[1];
        return 0;
    }

    //// ALGORYTM

    cout << "START" << endl;

    clock_t beginTime = clock();

    int indexOfFinishNode = getIndexOfNodeByName(namesOfNodes, argv[3]);
    int indexOfStartNode = getIndexOfNodeByName(namesOfNodes, argv[2]);
    int indexOfCurrentNode = indexOfStartNode;
    openEdges.push_back({ 0,0,indexOfStartNode,0 });//dodaje poczatek grafu to otwarych krawedzi

      //Zmienne CUDA
    int* finishX_D;
    int* finishY_D;
    int* numberOfPoints_D;
    hipMalloc((void**)&finishX_D, sizeof(int));
    hipMalloc((void**)&finishY_D, sizeof(int));
    hipMalloc((void**)&numberOfPoints_D, sizeof(int));
    hipMemcpy(finishX_D, &coords[indexOfFinishNode].x, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(finishY_D, &coords[indexOfFinishNode].y, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(numberOfPoints_D, &nodes.front().numberOfEdges, sizeof(int), hipMemcpyHostToDevice);


    double* heuristicValue_D;
    int* coordsX_D;
    int* coordsY_D;
    hipMalloc((void**)&heuristicValue_D, sizeof(double) * nodes.front().numberOfEdges);
    hipMalloc((void**)&coordsX_D, sizeof(int) * nodes.front().numberOfEdges);
    hipMalloc((void**)&coordsY_D, sizeof(int) * nodes.front().numberOfEdges);

    do
    {
        checkedEdges.push_back(openEdges.front());//dorzucam krawedz do sprawdzonych

        Node currentNode = nodes.at(indexOfCurrentNode);
        double weightFromStartOfCurrentNode = getWeightFromStartForNode(checkedEdges, indexOfCurrentNode); //pobieram rzeczywista wage do aktualnie wybranego node'a

        //CUDA
        double* heuristicValue = new double[currentNode.numberOfEdges];
        int* coordsX = new int[currentNode.numberOfEdges];
        int* coordsY = new int[currentNode.numberOfEdges];
        for (int i = 0; i < currentNode.numberOfEdges; i++)
        {
            coordsX[i] = coords[currentNode.idsOfNeighbor[i]].x;
            coordsY[i] = coords[currentNode.idsOfNeighbor[i]].y;
        }

        hipMemcpy(heuristicValue_D, heuristicValue, sizeof(double) * currentNode.numberOfEdges, hipMemcpyHostToDevice);
        hipMemcpy(coordsX_D, coordsX, sizeof(int) * currentNode.numberOfEdges, hipMemcpyHostToDevice);
        hipMemcpy(coordsY_D, coordsY, sizeof(int) * currentNode.numberOfEdges, hipMemcpyHostToDevice);


        calcHeuristicValues << <1, 500 >> > (coordsX_D, coordsY_D, heuristicValue_D, finishX_D, finishY_D, numberOfPoints_D);
        hipDeviceSynchronize();
        hipMemcpy(heuristicValue, heuristicValue_D, sizeof(double) * currentNode.numberOfEdges, hipMemcpyDeviceToHost);
        //

        for (int i = 0; i < currentNode.numberOfEdges; i++)
        {
            Edge tmpEdge;//tworze tymczasowa krawedz
            tmpEdge.idOfPreviusNode = indexOfCurrentNode;//krawedzi prowadzi od akutalnego node'a
            tmpEdge.idOfNode = currentNode.idsOfNeighbor[i];//do sasiada aktualnego node'a
            tmpEdge.weightFromStart = weightFromStartOfCurrentNode + currentNode.weightOfEdges[i];//obliczam rzeczywista wage od startu do sasiada
            tmpEdge.weight = tmpEdge.weightFromStart + calcHeuristicValue(coords[tmpEdge.idOfNode].x, coords[tmpEdge.idOfNode].y, coords[indexOfFinishNode].x, coords[indexOfFinishNode].y);

            if (isEdgeIsOpened(openEdges, tmpEdge.idOfNode)) //sprawdzam czy ta krawedzi jest juz w wektorze otwartych
            {
                int iterator = getIndexOfEdge(openEdges, tmpEdge.idOfNode);//jak tak to sprawdzam gdzie jest

                if (openEdges.at(iterator).weight > tmpEdge.weight)//jezeli jej waga w wektorze jest wieksza to podmieniam ja na tanszy odpowiednik
                {
                    openEdges.at(iterator) = tmpEdge;
                }
            }
            else if (isEdgeIsOpened(checkedEdges, tmpEdge.idOfNode)) //sprawdzam czy ta krawedzi jest juz w wektorze sprawdzonych
            {
                int iterator = getIndexOfEdge(checkedEdges, tmpEdge.idOfNode);//jak tak to sprawdzam gdzie jest

                if (checkedEdges.at(iterator).weight > tmpEdge.weight)//jezeli jej waga w wektorze jest wieksza to podmieniam ja na tanszy odpowiednik
                {
                    checkedEdges.at(iterator) = tmpEdge;
                }
            }
            else
            {
                openEdges.push_back(tmpEdge);
            }
        }


        openEdges.erase(openEdges.begin());//usuwam sprawdzona krawedz z wektora
        sort(openEdges.begin(), openEdges.end());//sortuje krawedzie do odwiedzenia

        indexOfCurrentNode = openEdges.front().idOfNode;//biore id najtanszej krawedzi
        if (indexOfCurrentNode == indexOfFinishNode)//sprawdzam czy nie jest ona celem 
        {
            checkedEdges.push_back(openEdges.front());//jak tak to dodaje do odwiedzonych
        }

    } while (indexOfCurrentNode != indexOfFinishNode);//wykonuje petle dopoki nie znajdzie celowej krawedzi

    clock_t endTime = clock();
    double elapsed_secs = double(endTime - beginTime) / CLOCKS_PER_SEC;
    cout << "Execution time: " << elapsed_secs << endl;

    Edge currentEdge = checkedEdges.back();//tymczasowa krawedz do doczytania wyniku
    cout << currentEdge.weightFromStart << endl;;

    cout << namesOfNodes.at(indexOfFinishNode) << endl;

    do
    {
        cout << namesOfNodes.at(currentEdge.idOfPreviusNode) << endl;//wyswietam nazwe poprzednika
        int idOfPreviusNode = currentEdge.idOfPreviusNode;
        currentEdge = checkedEdges.at(getIndexOfEdge(checkedEdges, idOfPreviusNode));
    } while (currentEdge.idOfNode != indexOfStartNode);//dopoki nie spotkam sie z pierwszym




    ////WYSWIETLANIE WSZYSTKIEGO

    //cout << numberOfNodes << endl;

    //for (int i = 0; i < numberOfNodes; i++)
    //{
    //    cout << nodes.at(i).name << " " << nodes.at(i).coord.x << " " << nodes.at(i).coord.y << " " << nodes.at(i).numberOfEdges;
    //    for (int j = 0; j < nodes.at(i).numberOfEdges; j++)
    //    {
    //        cout << " " << nodes.at(i).idsOfNeighbor[j] << " ";
    //        cout << nodes.at(i).weightOfEdges[j];
    //    }
    //    cout << " " << endl;
    //}

}

int getIndexOfNodeByName(vector<string> namesOfNodes, string name)
{
    auto iterator = find(namesOfNodes.begin(), namesOfNodes.end(), name);

    if (iterator != namesOfNodes.end())
    {
        int index = iterator - namesOfNodes.begin();
        return index;
    }
}

double calcHeuristicValue(int x1, int y1)
{
    return calcHeuristicValue(x1, y1, 0, 0);
}

double calcHeuristicValue(int x1, int y1, int x2, int y2)
{
    return sqrt(pow(((double)x2 - (double)x1), 2) + pow(((double)y2 - (double)y1), 2));
}

double getWeightFromStartForNode(vector<Edge> edges, int idOfNode)
{
    for (int i = 0; i < edges.size(); i++)
    {
        if (edges.at(i).idOfNode == idOfNode) return edges.at(i).weightFromStart;
    }

    return 0;
}

bool isEdgeIsOpened(vector<Edge> edges, int idOfNode)
{
    for (int i = 0; i < edges.size(); i++)
    {
        if (edges.at(i).idOfNode == idOfNode) return true;
    }

    return false;
}

int getIndexOfEdge(vector<Edge> edges, int idOfNode)
{
    for (int i = 0; i < edges.size(); i++)
    {
        if (edges.at(i).idOfNode == idOfNode) return i;
    }

    return 0;
}

__global__ void calcHeuristicValues(int* x, int* y, double* c, int* finishX, int* finishY, int* numberOfPoints)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < *numberOfPoints)
    {
        c[i] = sqrt(pow(((double)x[i] - (double)*finishX), 2) + pow(((double)y[i] - (double)*finishY), 2));

    }
}
